#include "hip/hip_runtime.h"

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include<hip/hip_runtime.h>
#include ""
#include <hip/device_functions.h>
#include<hip/hip_runtime.h>
#include "PAGNClumpyCloudFinderGPU.hpp"

namespace agn
{
	__global__ void main_algorithm_gpu(sPosition* cloud_d, phys_float R, phys_float x, phys_float y, phys_float z, phys_float omega_x, phys_float omega_y, phys_float omega_z, phys_size* index)
	{
		// we need the index of the current cloud
		auto indexCloud = blockIdx.x * blockDim.x + threadIdx.x;

		//auto indexCloud=0;

		//---ALGORITHM STARTS HERE--------------------------------------------------------------------------

		//initialy we get the vector from the photon
		//to the cloud: r_cloud - r_photon
		auto x_photonToCloud{ cloud_d[indexCloud].x - x };
		auto y_photonToCloud{ cloud_d[indexCloud].y - y };
		auto z_photonToCloud{ cloud_d[indexCloud].z - z };

		// we need the dot product to get the angle theta(see below)
		auto xyz_photonToCloud_Times_Omega = x_photonToCloud * omega_x + y_photonToCloud * omega_y + z_photonToCloud * omega_z;

		// we need the norm of the vectors
		auto r_photonToCloud = ::sqrt(x_photonToCloud * x_photonToCloud + y_photonToCloud * y_photonToCloud + z_photonToCloud * z_photonToCloud);
		auto omega = ::sqrt(omega_x * omega_x + omega_y * omega_y + omega_z * omega_z);

		// now we can get the angle between the radius vector of the cloud
		// and the dicrection(omega) of the photon
		auto theta = std::acos(xyz_photonToCloud_Times_Omega / (r_photonToCloud * omega));

		// this variable has a special meaning in this algorithm:
		// if the cloud is intersectec by the photon trajectory
		// then its value is the distance from the cloud to the photon
		// otherwise it is the max value the corresponding type
		phys_size distanceCloudToPhoton_here{ ULLONG_MAX };

		// only if the cloud is not "behind" the photon we calculate the distance
		// from the center of the cloud to the trajectory of the photon
		if (theta < (3.141592653589 / 2.0))
		{
			// this is the distance from the center of the cloud to
			// the photon trajectory
			auto distance_value = r_photonToCloud * ::sin(theta);

			// and only if the distance value is less than its radius
			// we set the variable distance_here to the value of
			// the distance from the photon to the cloud center type
			// casted to the corresponding interger value in order
			// to be able to use cuda atomic operations below
			if (distance_value < R)
			{
				distanceCloudToPhoton_here = __double2ull_rn(r_photonToCloud);
			}
			else
				return;
		}
		else
			return;

		//--- ALGORITHM FINISHES HERE--------------------------------------------------------------------------

		// at this point we have the distance to the photon from
		// the cloud, it will be THE MAX_VAL if the current cloud is not
		// intersected by the photon trajectory, otherwise it will be the real
		// distance BUT IN UNSIGNED LONG LONG-TYPE!!!

		// THE REMAINING PART OF THE CODE CONCERNS THE FINDING OF
		// THE FIRST CLOUD THAT WILL BE INTERSECTED BY THE PHOTON
		// IF THERE IS AT LEAST ONE IN THE CORRESPONDING THREAD BLOCK

		// now we need a shared variable that will represent the distance
		// to the best candidate, among all the clouds in the current block
		__shared__ phys_size distanceToBestCandidate;

		// we set the value of the distance to the max possible value
		// width the corresponding type
		distanceToBestCandidate = ULLONG_MAX;

		// we wait until all threads get to this point and the value
		// of distance_here is properly set
		__syncthreads();

		// if the current cloud is intersected by the photon's trajectory
		// then its distance to the photon will be less than the init value
		// ie less than the ULLONG_MAX in our algorithm
		if (distanceCloudToPhoton_here < ULLONG_MAX)
		{
			// because the value of distance will be compared with the other
			// values of others threads, one by one we use atomicMin()
			::atomicMin(&distanceToBestCandidate, distanceCloudToPhoton_here);
		}

		// we wait untill the final value of distanceToBestCandidate is set,
		// this can be one of two values: the distance from the photon to the first
		// cloud that it will intersect or the max value initially set, if the photon
		// will not intersect any cloud at all
		__syncthreads();

		// now we can comprare if the distance of the current cloud to
		// the photon is the distance from the photon to the first
		// cloud that will be intersected. on the other hand it is possible
		// that the photon wont intersect any cloud, in that case
		// the variable distanceToBestCandidate will be the initial, ie the max value
		// so its important to check that condition also
		// to sumarise: only if there is a cloud that will be intersected
		// by the photon first, and this thread corresponds to that cloud
		// we set the index of that cloud in the general array of clouds
		// to the block of candidates

		if (distanceToBestCandidate != ULLONG_MAX && distanceToBestCandidate == distanceCloudToPhoton_here)
		{
			index[blockIdx.x] = indexCloud;
		}
	}

	__global__ void foo2()
	{
		int i = threadIdx.x + 1 * 3;
	}

	void callMainAlgorithGPU(sPosition* cloud_d, phys_float R,
		phys_float x, phys_float y, phys_float z,
		phys_float omega_x, phys_float omega_y, phys_float omega_z,
		phys_size* indexes_d,
		phys_size NUM_OF_BLOCKS, phys_size NUM_OF_THREADS_PER_BLOCK, hipStream_t s)
	{
		main_algorithm_gpu << <NUM_OF_BLOCKS, NUM_OF_THREADS_PER_BLOCK >> > (cloud_d, R, x, y, z, omega_x, omega_y, omega_z, indexes_d);
		//foo2 << < NUM_OF_BLOCKS, NUM_OF_THREADS_PER_BLOCK >> > ();
	}

}// namespace agn