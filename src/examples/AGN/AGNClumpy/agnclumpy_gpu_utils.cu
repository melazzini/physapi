#include "hip/hip_runtime.h"

#define CUDA_API_PER_THREAD_DEFAULT_STREAM

#include<hip/hip_runtime.h>
#include ""
#include <hip/device_functions.h>
#include<hip/hip_runtime.h>
#include "PAGNClumpyCloudFinderGPU.hpp"

namespace agn
{
	__global__ void foo2()
	{
		int i = threadIdx.x + 1 * 3;
	}

	void callMainAlgorithGPU(sPosition* cloud_d, phys_float R,
		phys_float x, phys_float y, phys_float z,
		phys_float omega_x, phys_float omega_y, phys_float omega_z,
		phys_size* indexes_d,
		phys_size NUM_OF_BLOCKS, phys_size NUM_OF_THREADS_PER_BLOCK, hipStream_t s)
	{
		foo2 << < NUM_OF_BLOCKS, NUM_OF_THREADS_PER_BLOCK >> > ();
	}

}// namespace agn